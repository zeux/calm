#include "hip/hip_runtime.h"
#include "model.h"

#include <assert.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>

#include <hip/hip_cooperative_groups.h>

#include "helpers.cuh"

#define CUDA_CHECK(x)                                                                                    \
	do {                                                                                                 \
		hipError_t err = x;                                                                             \
		if (err != hipSuccess) {                                                                        \
			fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, __FILE__, __LINE__, \
			        hipGetErrorString(err), hipGetErrorName(err), err);                                \
			abort();                                                                                     \
		}                                                                                                \
	} while (0)

#define PROF_TOKEN(bytes) ((0xCDAFull << 48) | (bytes))

template <typename T>
struct CoopLayer {
	float* rms_att_weight;
	T* wq;
	T* wk;
	T* wv;
	T* wo;
	float* bqkv;

	float* rms_ffn_weight;
	T* moegate;
	T* w1;
	T* w2;
	T* w3;
};

static int ngpus;
static unsigned int* xbarrier;

static hipStream_t stream;

static int coopsms;

static __constant__ CoopLayer<void> cooplayers[MAX_LAYERS];

static uint64_t* coopperf;
static uint64_t coopperfbw[16];
static int coopruns;

static void* cuda_devicecopy(void* host, size_t size) {
	void* device = NULL;
	CUDA_CHECK(hipMalloc(&device, size));
	CUDA_CHECK(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice));
	return device;
}

static void* cuda_devicealloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipMalloc(&ptr, size));
	return ptr;
}

static void* cuda_hostalloc(size_t size) {
	void* ptr = NULL;
	CUDA_CHECK(hipHostAlloc(&ptr, size, 0));
	return ptr;
}

extern "C" void init_cuda() {
	const char* ng = getenv("CALM_NG");
	if (ng && atoi(ng)) {
		ngpus = atoi(ng);
		printf("# CUDA: Using %d GPUs (experimental)\n", ngpus);

		for (int i = 0; i < ngpus; ++i) {
			CUDA_CHECK(hipSetDevice(i));
			for (int j = 0; j < ngpus; ++j) {
				if (i != j) {
					CUDA_CHECK(hipDeviceEnablePeerAccess(j, 0));
				}
			}
		}

		CUDA_CHECK(hipSetDevice(0));

		xbarrier = (unsigned int*)cuda_devicealloc(sizeof(unsigned int) * ngpus);
		CUDA_CHECK(hipMemset(xbarrier, 0, sizeof(unsigned int) * ngpus));
	}
}

extern "C" void* upload_cuda(void* host, size_t size) {
	if (ngpus) {
		void* res = NULL;
		CUDA_CHECK(hipMallocManaged(&res, size));
		memcpy(res, host, size);

		CUDA_CHECK(hipMemAdvise(res, size, hipMemAdviseSetReadMostly, 0));
		for (int i = 0; i < ngpus; i++) {
			CUDA_CHECK(hipMemPrefetchAsync(res, size, i));
		}
		return res;
	} else {
		return cuda_devicecopy(host, size);
	}
}

extern "C" void prepare_cuda(struct Transformer* transformer) {
	struct Config* config = &transformer->config;
	struct Weights* weights = &transformer->weights;
	struct RunState* state = &transformer->state;

	hipDeviceProp_t devprop = {};
	CUDA_CHECK(hipGetDeviceProperties(&devprop, 0));
	assert(devprop.cooperativeLaunch);

	printf("# CUDA: %s, compute %d.%d, %d SMs, %.1f GiB, peak bandwidth %.0f GB/s (ECC %d)\n",
	       devprop.name, devprop.major, devprop.minor, devprop.multiProcessorCount,
	       (double)devprop.totalGlobalMem / (1024 * 1024 * 1024),
	       (double)devprop.memoryClockRate * (devprop.memoryBusWidth / 8) * 2 / 1e6, devprop.ECCEnabled);

	coopsms = devprop.multiProcessorCount;

	if (getenv("CUDA_INJECTION64_PATH")) {
		coopperf = (uint64_t*)cuda_devicealloc(sizeof(uint64_t) * 16);
		CUDA_CHECK(hipMemset(coopperf, 0, sizeof(uint64_t) * 16));
	}

	CUDA_CHECK(hipStreamCreate(&stream));

	int dim = config->dim;
	int hidden_dim = config->hidden_dim;
	int q_dim = config->head_dim * config->n_heads;
	int kv_dim = config->head_dim * config->n_kv_heads;

	state->x = (float*)cuda_devicealloc(dim * sizeof(float));
	state->hb = (float*)cuda_devicealloc(hidden_dim * sizeof(float));
	state->he = (float*)cuda_devicealloc(config->n_experts_ac * hidden_dim * sizeof(float));
	state->q = (float*)cuda_devicealloc(q_dim * sizeof(float));
	state->att = (float*)cuda_devicealloc(config->n_heads * config->seq_len * 2 * sizeof(float));

	assert(state->kvbits == 8 || state->kvbits == 16);
	state->key_cache = cuda_devicealloc((size_t)config->n_layers * config->seq_len * kv_dim * (state->kvbits / 8));
	state->value_cache = cuda_devicealloc((size_t)config->n_layers * config->seq_len * kv_dim * (state->kvbits / 8));

	// logits are going to be read by the host so we just allocate them in host and write to host directly
	state->logits = (float*)cuda_hostalloc(config->vocab_size * sizeof(float));

	CoopLayer<void> layers[MAX_LAYERS];
	for (int l = 0; l < config->n_layers; ++l) {
		layers[l].rms_att_weight = weights->rms_att_weight[l];
		layers[l].wq = weights->wq[l];
		layers[l].wk = weights->wk[l];
		layers[l].wv = weights->wv[l];
		layers[l].wo = weights->wo[l];
		layers[l].bqkv = weights->bqkv[l];

		layers[l].rms_ffn_weight = weights->rms_ffn_weight[l];
		layers[l].moegate = weights->moegate[l];
		layers[l].w1 = weights->w1[l];
		layers[l].w2 = weights->w2[l];
		layers[l].w3 = weights->w3[l];
	}

	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cooplayers), layers, sizeof(layers)));

	if (ngpus) {
		for (int i = 1; i < ngpus; ++i) {
			CUDA_CHECK(hipSetDevice(i));
			CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cooplayers), layers, sizeof(layers)));
		}
		CUDA_CHECK(hipSetDevice(0));
	}
}

template <typename T>
__device__ inline float embed(T* weight, int idx) {
	return float(weight[idx]);
}

__device__ inline float embed(uint32_t* weight, int idx) {
	return gf4_ff(weight[idx / 8], idx % 8);
}

template <typename T>
__global__ static void kernel_embed(float* o, T* weight, int token, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	assert(i < n);

	o[i] = embed(weight, token * n + i);
}

template <typename KVT>
__global__ static void kernel_rotate_sink(uint64_t, int kvd, KVT* key_cache, int head_dim, int kv_sink, float theta_log2, int seq_len, int rotary_dim) {
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	assert(i < kv_sink * kvd);

	int l = blockIdx.y;

	int j_head = i % head_dim;
	float freq = j_head >= rotary_dim ? 0.f : exp2f(-theta_log2 * (float)j_head / (float)rotary_dim);

	// rotate sink tokens forward to keep pace with non-sink tokens
	float fcr, fci;
	sincosf(freq, &fci, &fcr);

	size_t loff = (size_t)l * seq_len * kvd;
	KVT* kb = key_cache + loff;

	// note: k layout is transposed / tiled to improve attn_score performance
	int t = i / kvd;
	int k = i % kvd;
	int o = t * 16 + seq_len * (k / 16) * 16 + (k % 16);

	float v0 = float(kb[o + 0]);
	float v1 = float(kb[o + 1]);

	float r0 = v0 * fcr - v1 * fci;
	float r1 = v0 * fci + v1 * fcr;

	kb[o + 0] = KVT(r0);
	kb[o + 1] = KVT(r1);
}

__device__ inline float gelu(float x) {
	return 0.5f * x * (1.0f + tanhf(0.797885f * (x + 0.044715f * x * x * x)));
}

__device__ inline float silu(float x) {
	return x / (1.0f + expf(-x));
}

__device__ static void moe_gate_warp(float* moe_weights, int* moe_experts, float* weights, int experts, int active) {
	int i = threadIdx.x;

	// (unscaled) softmax across experts
	float w = (i < experts) ? weights[i] : -FLT_MAX;
	float max_val = warpreduce_max(w);
	w = expf(w - max_val);

	// weight in top 24 bits, index in bottom 8
	int wi = (__float_as_int(w) & 0xffffff00) | i;

	// top k within warp
	float sumw = 0.f;
	int acti = -1;

	for (int k = 0; k < active; ++k) {
		int maxi = warpreduce_maxi(wi);

		sumw += __int_as_float(maxi);

		// keeps top weight in thread k, clears weight for thread with max thread to avoid re-selection
		acti = (i == k) ? maxi : acti;
		wi = (wi == maxi) ? 0 : wi;
	}

	// write normalized weights
	if (i < active) {
		assert(acti >= 0);

		moe_experts[i] = acti & 0xff;
		moe_weights[i] = __int_as_float(acti) / sumw;
	}
}

union half4 {
	float2 g;
	half h[4];
};

__device__ inline float4 attn_load4(half* p) {
	half4 h = *(half4*)p;
	return {__half2float(h.h[0]), __half2float(h.h[1]), __half2float(h.h[2]), __half2float(h.h[3])};
}

__device__ inline float4 attn_load4(__hip_fp8_e5m2_fnuz* p) {
	return fp8x4_e5m2_ff(*(__hip_fp8x4_e5m2_fnuz*)p);
}

template <typename KVT>
__device__ inline float attn_score(KVT* kht, float* qh, int head_dim, int seq_len, int t, int off) {
	float score = 0.0f;
	for (int j = 0; j < head_dim; j += 16) {
		float4 kk = attn_load4(&kht[j * seq_len + t * 16 + off]);
		float4 qq = *(float4*)&qh[j + off];
		score += kk.x * qq.x;
		score += kk.y * qq.y;
		score += kk.z * qq.z;
		score += kk.w * qq.w;
	}

	return score;
}

template <typename KVT>
__device__ inline float attn_warpdot(KVT* val, float* atth, int kv_len) {
	int kv_len4 = kv_len & ~3;
	int lane = threadIdx.x % warpSize;

	float res = 0.0f;
	float sum = 0.0f;
	for (int t = lane * 4; t < kv_len4; t += warpSize * 4) {
		float4 vv = attn_load4(&val[t]);
		float4 aa = *(float4*)&atth[t];
		res += vv.x * aa.x;
		res += vv.y * aa.y;
		res += vv.z * aa.z;
		res += vv.w * aa.w;
		sum += aa.x + aa.y + aa.z + aa.w;
	}

	if (kv_len4 + lane < kv_len) {
		float a = atth[kv_len4 + lane];
		res += a * float(val[kv_len4 + lane]);
		sum += a;
	}

	res = warpreduce_sum(res);
	sum = warpreduce_sum(sum);

	return res / sum;
}

__device__ static void softmax(float* xout, float* x, int size) {
	int i = threadIdx.x;

	// find max value per thread (for numerical stability)
	float max_val = -FLT_MAX;
	for (int j = i; j < size; j += blockDim.x) {
		max_val = max(max_val, x[j]);
	}

	// max across threads in block
	max_val = blockreduce_max(max_val);

	// exp per thread
	for (int j = i; j < size; j += blockDim.x) {
		xout[j] = expf(x[j] - max_val);
	}
}

__device__ static float rmsnorm(half* o, float* x, float* weight, int size, float eps, bool ln) {
	int i = threadIdx.x;
	int blockSize = blockDim.x;

	float mean = 0.0f;
	if (ln) {
		// calculate sum (per thread)
		float sum = 0.0f;
		for (int j = i; j < size; j += blockSize) {
			sum += x[j];
		}

		// sum across threads in block
		mean = blockreduce_sum(sum) / size;
	}

	// calculate sum of squares (per thread)
	float ss = 0.0f;
	for (int j = i; j < size; j += blockSize) {
		float v = x[j] - mean;
		ss += v * v;
		o[j] = v * weight[j];
	}

	// sum across threads in block
	ss = blockreduce_sum(ss);

	// caller is responsible for normalization
	return rsqrtf(ss / size + eps);
}

__device__ static void syncgrid() {
	volatile unsigned int* barrier = &cooperative_groups::details::get_grid_workspace()->barrier;

	if (threadIdx.x == 0) {
		unsigned int nb = 1;
		if (blockIdx.x == 0) {
			nb = 0x80000000 - (gridDim.x - 1);
		}

		unsigned int old_arrive;
		asm volatile("atom.add.release.gpu.u32 %0,[%1],%2;" : "=r"(old_arrive) : _CG_ASM_PTR_CONSTRAINT(barrier), "r"(nb) : "memory");

		unsigned int current_arrive;
		do {
			asm volatile("ld.acquire.gpu.u32 %0,[%1];" : "=r"(current_arrive) : _CG_ASM_PTR_CONSTRAINT(barrier) : "memory");
		} while (((old_arrive ^ current_arrive) & 0x80000000) == 0);
	}

	__syncthreads();
}

__device__ unsigned int load_arrived(volatile unsigned int* arrived) {
	unsigned int result;
	asm volatile("ld.acquire.sys.global.u32 %0, [%1];"
	             : "=r"(result)
	             : "l"(arrived)
	             : "memory");
	return result;
}

__device__ void store_arrived(volatile unsigned int* arrived, unsigned int val) {
	asm volatile(
	    "st.release.sys.global.u32 [%1], %0;" ::"r"(val) "l"(arrived)
	    : "memory");
}

__device__ static unsigned int syncgpus_arrive(int gpu, int n_gpus, volatile unsigned int* xbarrier) {
	unsigned int token = 0;
	if (blockIdx.x == 0 && threadIdx.x == 0) {
		token = xbarrier[gpu] + 1;
		store_arrived(&xbarrier[gpu], token);
	}
	return token;
}

__device__ static unsigned int syncgpus_wait(unsigned int mask, int n_gpus, volatile unsigned int* xbarrier, unsigned int token) {
	if (blockIdx.x == 0 && threadIdx.x == 0) {
		for (int j = 0; j < n_gpus; ++j) {
			if (mask & (1 << j)) {
				while (load_arrived(&xbarrier[j]) != token)
					;
			}
		}
	}
}

__device__ static unsigned int syncgpus(int gpu, int n_gpus, volatile unsigned int* xbarrier) {
	unsigned int token = syncgpus_arrive(gpu, n_gpus, xbarrier);
	syncgpus_wait(0xffffffff, n_gpus, xbarrier, token);
}

template <typename T, typename KVT>
struct CoopArgs {
	uint64_t bw;
	uint64_t* perfstats;

	unsigned int* xbarrier;
	int gpu;
	int n_gpus;

	float* x;
	__half* hb;
	float* q;
	__half* ab;
	float* att;

	KVT* key_cache;
	KVT* val_cache;

	int n_layers;

	int dim;
	int hidden_dim;
	int head_dim;
	int n_heads;
	int n_kv_heads;
	int n_experts;
	int n_experts_ac;
	int seq_len;
	int rotary_dim;

	bool norm_ln;
	bool act_gelu;

	int kv_len;
	int kv_pos;
	int pos;

	float norm_eps;
	float theta_log2;
};

__device__ static void coopstage(uint64_t* stats, int stage) {
	__shared__ uint64_t lastt;

	if (stats && blockIdx.x == 0 && threadIdx.x == 0) {
		uint64_t t;
		asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(t));

		if (stage >= 0) {
			stats[stage] += t - lastt;
		}
		lastt = t;
	}
}

template <typename T, typename KVT>
__global__ __launch_bounds__(1024, 1) static void kernel_forward(const __grid_constant__ CoopArgs<T, KVT> args) {
	extern __shared__ half xs[];
	__shared__ float rmsscale;

	__shared__ float moe_weights[32];
	__shared__ int moe_experts[32];

	int dim = args.dim;
	int hidden_dim = args.hidden_dim;
	int head_dim = args.head_dim;

	int kv_mul = args.n_heads / args.n_kv_heads;
	int q_dim = args.head_dim * args.n_heads;
	int kv_dim = args.head_dim * args.n_kv_heads;

	const int IK = 4; // K consecutive warps per block, groups of K are interleaved across SMs for better work distribution
	int io = blockIdx.x * IK + (threadIdx.x / warpSize % IK) + gridDim.x * IK * (threadIdx.x / warpSize / IK);
	int ib = (gridDim.x * blockDim.x) / warpSize;

	// for multigpu-friendly work we can use go..gb split instead which partitions work evenly across GPUs (assuming IK=1)
	int go = blockIdx.x + gridDim.x * (args.gpu + args.n_gpus * (threadIdx.x / warpSize));
	int gb = args.n_gpus * gridDim.x * (blockDim.x / warpSize);

	// dummy moe weights for non-moe models; will be overwritten by moe gate
	moe_weights[0] = 1.f;
	moe_experts[0] = 0;

	coopstage(args.perfstats, -1); // init timing

	static __device__ int badsoftmax = 0;

	for (int l = 0; l < args.n_layers; ++l) {
		const CoopLayer<T>* L = (const CoopLayer<T>*)&cooplayers[l];

		if (blockIdx.x == 0 && threadIdx.x < warpSize) {
			badsoftmax = 0;
		}

		// pre-attention rmsnorm (into shared memory)
		if (args.gpu == 0)
		rmsscale = rmsnorm(xs, args.x, L->rms_att_weight, dim, args.norm_eps, args.norm_ln);

		size_t loff = (size_t)l * args.seq_len * kv_dim; // kv cache layer offset for convenience
		KVT* keyb = args.key_cache + loff;
		KVT* valb = args.val_cache + loff;

		// qkv matmul + RoPE encoding + update KV cache
		if (args.gpu == 0)
		for (int j = io * 2; j < q_dim + kv_dim * 2; j += ib * 2) {
			T* w = j < q_dim ? L->wq : (j < q_dim + kv_dim ? L->wk : L->wv);
			int k = j < q_dim ? j : (j < q_dim + kv_dim ? j - q_dim : j - q_dim - kv_dim);

			float v0 = matmul_warppar(xs, w, k + 0, dim) * rmsscale;
			float v1 = matmul_warppar(xs, w, k + 1, dim) * rmsscale;

			if (L->bqkv) {
				v0 += L->bqkv[j + 0];
				v1 += L->bqkv[j + 1];
			}

			if (threadIdx.x % warpSize == 0) {
				int j_head = j % head_dim;
				float freq = j_head >= args.rotary_dim ? 0.f : exp2f(-args.theta_log2 * (float)j_head / (float)args.rotary_dim);
				float fcr, fci;
				sincosf(args.pos * freq, &fci, &fcr);

				if (j < q_dim) {
					args.q[k + 0] = v0 * fcr - v1 * fci;
					args.q[k + 1] = v0 * fci + v1 * fcr;
				} else if (j < q_dim + kv_dim) {
					// note: k layout is transposed / tiled to improve attn_score performance
					int off = args.kv_pos * 16 + args.seq_len * (k / 16) * 16 + (k % 16);
					keyb[off + 0] = KVT(v0 * fcr - v1 * fci);
					keyb[off + 1] = KVT(v0 * fci + v1 * fcr);
				} else {
					// note: v layout is transposed (we store all positions for a given head contiguously) to improve attn_mix performance
					valb[args.kv_pos + args.seq_len * (k + 0)] = KVT(v0);
					valb[args.kv_pos + args.seq_len * (k + 1)] = KVT(v1);
				}
			}
		}

		syncgrid();
		coopstage(args.perfstats, 0);

		// attention score
		int kv_lent = (args.kv_len + 7) / 8;

		if (args.gpu == 0)
		for (int j = io; j < kv_lent * args.n_heads; j += ib) {
			int h = j % args.n_heads;
			int kvh = h / kv_mul;
			int t = (j / args.n_heads) * 8 + (threadIdx.x % warpSize) / 4;

			unsigned active = __ballot_sync(0xffffffff, t < args.kv_len);

			if (t < args.kv_len) {
				float* qh = args.q + h * head_dim;
				KVT* kh = keyb + kvh * head_dim * args.seq_len;
				float* atth = args.att + h * args.seq_len * 2;

				float score = attn_score(kh, qh, head_dim, args.seq_len, t, 4 * (threadIdx.x % 4));

				// reduce score across threads in warp; every 4 threads are processing the same output score
				score += __shfl_xor_sync(active, score, 2);
				score += __shfl_xor_sync(active, score, 1);
				score /= sqrtf(head_dim);

				atth[t] = expf(score);
				atth[t + args.seq_len] = score;

				// to reduce latency we prefer computing softmax without the numeric stabilization, which is safe if all inputs are small
				if (fabsf(score) > 40) {
					badsoftmax = 1;
				}
			}
		}

		syncgrid();
		coopstage(args.perfstats, 1);

		if (args.gpu == 0 && badsoftmax) {
			// attention softmax
			if (blockIdx.x < args.n_heads) {
				int h = blockIdx.x;
				float* atth = args.att + h * args.seq_len * 2;

				softmax(atth, atth + args.seq_len, args.kv_len);
			}

			syncgrid();
			coopstage(args.perfstats, 2);
		}

		// attention mix
		if (args.gpu == 0)
		for (int j = io; j < q_dim; j += ib) {
			int h = j / head_dim;
			int kvh = h / kv_mul;
			int j_head = j % head_dim;

			float* atth = args.att + h * args.seq_len * 2;
			KVT* vh = valb + kvh * head_dim * args.seq_len;
			KVT* val = vh + j_head * args.seq_len;

			float res = attn_warpdot(val, atth, args.kv_len);

			if (threadIdx.x % warpSize == 0) {
				args.ab[j] = res;
			}
		}

		syncgrid();
		coopstage(args.perfstats, 3);

		// attention output
		if (args.gpu == 0)
		for (int j = io; j < dim; j += ib) {
			float val = matmul_warppar(args.ab, L->wo, j, q_dim);

			if (threadIdx.x % warpSize == 0) {
				args.x[j] += val;
			}
		}

		syncgrid();
		syncgpus(args.gpu, args.n_gpus, args.xbarrier);
		syncgrid();
		coopstage(args.perfstats, 4);

		// post-attention rmsnorm (into shared memory)
		rmsscale = rmsnorm(xs, args.x, L->rms_ffn_weight, dim, args.norm_eps, args.norm_ln);

		// moegate
		if (args.n_experts) {
			__shared__ float exp[32];
			int j = threadIdx.x / warpSize;

			if (j < args.n_experts) {
				float val = matmul_warppar(xs, L->moegate, j, dim) * rmsscale;

				exp[j] = val;
			}

			__syncthreads();

			if (threadIdx.x < warpSize) {
				moe_gate_warp(moe_weights, moe_experts, exp, args.n_experts, args.n_experts_ac);
			}

			__syncthreads();
		}

		// important!!! careful with distribution/handling
		int e = args.gpu % args.n_experts_ac;

		// F.silu(self.w1(x)) * self.w3(x)
		for (int j = io; j < hidden_dim; j += ib) {
			int je = j + moe_experts[e] * hidden_dim;
			float v1 = matmul_warppar(xs, L->w1, je, dim) * rmsscale;
			float v3 = matmul_warppar(xs, L->w3, je, dim) * rmsscale;

			float val = (args.act_gelu ? gelu(v1) : silu(v1)) * v3;

			if (threadIdx.x % warpSize == 0) {
				args.hb[j + e * hidden_dim] = val;
			}
		}

		syncgrid();
		syncgpus(args.gpu, args.n_gpus, args.xbarrier);
		syncgrid();
		coopstage(args.perfstats, 5);

		// self.w2(...) + pre-rmsnorm residual
		if (args.gpu == 0)
		for (int j = io; j < dim; j += ib) {
			float val = 0.f;
			for (int e = 0; e < args.n_experts_ac; ++e) {
				int je = j + moe_experts[e] * dim;
				val += matmul_warppar(args.hb + e * hidden_dim, L->w2, je, hidden_dim) * moe_weights[e];
			}

			if (threadIdx.x % warpSize == 0) {
				args.x[j] += val;
			}
		}

		syncgrid();
		coopstage(args.perfstats, 6);
	}
}

template <typename T>
__global__ static void kernel_output(uint64_t, float* xout, float* x, T* w, float* rms_weight, int n, int d, float norm_eps, bool norm_ln) {
	extern __shared__ half xs[];

	float rmsscale = rmsnorm(xs, x, rms_weight, n, norm_eps, norm_ln);

	int io = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
	int ib = (gridDim.x * blockDim.x) / warpSize;

	for (int j = io; j < d; j += ib) {
		float val = matmul_warppar(xs, w, j, n) * rmsscale;

		// instead of writing one value per block, we transpose the values and write all results from first warp
		val = blocktranspose(val, 0.f);

		if (threadIdx.x < blockDim.x / warpSize) {
			xout[j + threadIdx.x] = val;
		}
	}
}

template <typename T, typename KVT>
static float* forward(struct Transformer* transformer, int token, int pos, unsigned flags) {
	struct Config* p = &transformer->config;
	struct Weights* w = &transformer->weights;
	struct RunState* s = &transformer->state;

	// a few convenience variables
	float* x = s->x;
	int dim = p->dim;
	int hidden_dim = p->hidden_dim;
	int kv_dim = p->head_dim * p->n_kv_heads;
	size_t dbits = w->dbits; // size_t prevents integer overflow in multiplications below

	// following "attention sinks" from StreamingLLM we keep the first few tokens in the KV cache as is
	int kv_sink = pos >= p->seq_len ? KV_SINKS : 0;
	int kv_pos = kv_sink + (pos - kv_sink) % (p->seq_len - kv_sink);
	int kv_len = pos >= p->seq_len ? p->seq_len : pos + 1;

	// ensure all dimensions are warp-aligned
	assert(dim % 32 == 0 && kv_dim % 32 == 0 && hidden_dim % 32 == 0);

	// copy the token embedding into x
	assert(token < p->vocab_size);
	kernel_embed<<<dim / 32, 32, 0, stream>>>(x, (T*)w->token_embedding_table, token, dim);

	// rotate sink tokens forward to keep pace with non-sink tokens
	if (kv_sink > 0) {
		kernel_rotate_sink<<<dim3(kv_sink * kv_dim / 64, p->n_layers), 32, 0, stream>>>(
			PROF_TOKEN(kv_sink * kv_dim * sizeof(KVT)), kv_dim, (KVT*)s->key_cache, p->head_dim, kv_sink, log2(p->rope_theta), p->seq_len, p->rotary_dim);
	}

	// forward all the layers
	size_t kvbw = p->n_kv_heads * p->head_dim * kv_len * sizeof(KVT) + p->n_heads * kv_len * sizeof(float);

	uint64_t bw = 0;
	bw += (dim + kv_dim * 2) * dim * dbits / 8; // QKV
	bw += kvbw * 2; // attn scoring and mixing
	bw += dim * dim * dbits / 8; // attn output
	bw += 3 * (hidden_dim * dim * dbits / 8) * max(p->n_experts_ac, 1); // MLP
	bw *= p->n_layers;

	coopruns++;
	coopperfbw[0] += (size_t)p->n_layers * ((dim + kv_dim * 2) * dim * dbits / 8); // QKV
	coopperfbw[1] += (size_t)p->n_layers * kvbw; // attn scoring
	coopperfbw[2] += 0; // attn softmax
	coopperfbw[3] += (size_t)p->n_layers * kvbw; // attn mixing
	coopperfbw[4] += (size_t)p->n_layers * (dim * dim * dbits / 8); // attn output
	coopperfbw[5] += (size_t)p->n_layers * (2 * (hidden_dim * dim * dbits / 8) * max(p->n_experts_ac, 1)); // MLP
	coopperfbw[6] += (size_t)p->n_layers * (1 * (hidden_dim * dim * dbits / 8) * max(p->n_experts_ac, 1)); // MLP

	CoopArgs<T, KVT> args = {
	    PROF_TOKEN(bw),
	    coopperf,
	    // multi-gpu state
	    xbarrier, 0, max(ngpus, 1),
	    // token state
	    x,
	    (half*)(p->n_experts ? s->he : s->hb),
	    s->q,
	    (half*)s->q,
	    s->att,
	    // key/value cache; note that layers are passed via cooplayers[]
	    (KVT*)s->key_cache,
	    (KVT*)s->value_cache,
	    // model dimensions
	    p->n_layers,
	    dim,
	    hidden_dim,
	    p->head_dim,
	    p->n_heads,
	    p->n_kv_heads,
	    p->n_experts,
	    max(p->n_experts_ac, 1),
	    p->seq_len,
	    p->rotary_dim,
	    // model configuration
	    p->norm_ln,
	    p->act_gelu,
	    // token position (and derived data)
	    kv_len,
	    kv_pos,
	    pos,
	    // model parameters
	    p->norm_eps,
	    log2(p->rope_theta),
	};
	void* argsp = &args;

	size_t coop_smem = dim * sizeof(half);

	CUDA_CHECK(hipFuncSetAttribute((void*)kernel_forward<T, KVT>, hipFuncAttributeMaxDynamicSharedMemorySize, coop_smem));
	CUDA_CHECK(hipLaunchCooperativeKernel((void*)kernel_forward<T, KVT>, coopsms, 1024, &argsp, coop_smem, stream));

	if (ngpus) {
		for (int i = 1; i < ngpus; i++) {
			args.perfstats = NULL;
			args.gpu = i;
			CUDA_CHECK(hipSetDevice(i));
			CUDA_CHECK(hipFuncSetAttribute((void*)kernel_forward<T, KVT>, hipFuncAttributeMaxDynamicSharedMemorySize, coop_smem));
			CUDA_CHECK(hipLaunchCooperativeKernel((void*)kernel_forward<T, KVT>, coopsms, 1024, &argsp, coop_smem));
		}
		CUDA_CHECK(hipSetDevice(0));
	}

	if (flags & FF_UPDATE_KV_ONLY) {
		// only update kv cache and don't output logits
		return NULL;
	}

	int output_blk = 32 * 32;
	int output_par = 1;
	CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&output_par, kernel_output<T>, output_blk, dim * sizeof(float)));

	// classifier into logits
	kernel_output<<<coopsms * output_par, output_blk, dim * sizeof(float), stream>>>(
	    PROF_TOKEN(p->vocab_size * dim * dbits / 8), s->logits, x, (T*)w->wcls, w->rms_final_weight, dim, p->vocab_size, p->norm_eps, p->norm_ln);

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipGetLastError()); // check for kernel launch errors; they might fail with OOM due to lazy kernel compilation

	return s->logits;
}

extern "C" float* forward_cuda(struct Transformer* transformer, int token, int pos, unsigned flags) {
#define CASE(dbits_, dtype, kvbits_, kvtype)                                          \
	if (transformer->weights.dbits == dbits_ && transformer->state.kvbits == kvbits_) \
	return forward<dtype, kvtype>(transformer, token, pos, flags)

	assert(ngpus > 0);
	assert(ngpus >= transformer->config.n_experts_ac);
	assert(ngpus % transformer->config.n_experts_ac == 0);

	// CASE(4, uint32_t, 8, __hip_fp8_e5m2_fnuz);
	// CASE(4, uint32_t, 16, __half);
	CASE(8, __hip_fp8_e5m2_fnuz, 8, __hip_fp8_e5m2_fnuz);
	CASE(8, __hip_fp8_e5m2_fnuz, 16, __half);
	// CASE(16, __half, 8, __hip_fp8_e5m2_fnuz);
	// CASE(16, __half, 16, __half);

	assert(!"Unsupported dbits/kvbits combination for CUDA: dbits must be 4, 8 or 16, kvbits must be 8 or 16");
	return NULL;

#undef CASE
}

extern "C" void perf_cuda() {
	if (coopperf == NULL || coopruns == 0)
		return;

	uint64_t hostperf[16] = {};
	CUDA_CHECK(hipMemcpy(hostperf, coopperf, sizeof(hostperf), hipMemcpyDeviceToHost));

	static const char* stagenames[16] = {
	    "matmul_qkv",
	    "attn_score",
	    "attn_softmax",
	    "attn_mix",
	    "matmul_attn",
	    "matmul_ffn_up",
	    "matmul_ffn_down",
	};

	double freq = 1e9;

	uint64_t total = 0;
	for (int stage = 0; stage < 16; ++stage) {
		total += hostperf[stage];
	}

	printf("\nkernel_forward breakdown (over %d runs, avg %.1f usec/run):\n",
	       coopruns, (double)total / (double)coopruns / freq * 1e6);

	for (int stage = 0; stage < 16; ++stage) {
		if (hostperf[stage] == 0)
			continue;

		uint64_t t = hostperf[stage];
		uint64_t tbw = coopperfbw[stage];

		printf("\t[%d] %16s: %4.1f%%; %8.1f usec/run, %6.1f GB/s\n",
		       stage, stagenames[stage],
		       (double)t / (double)total * 100,
		       (double)(t / coopruns) / freq * 1e6,
		       ((double)tbw / 1e9) / ((double)t / freq));
	}
}
